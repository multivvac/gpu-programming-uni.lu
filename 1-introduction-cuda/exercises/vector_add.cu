
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

void genNFP(float* h, int n) {
  srand(time(0));
  for (int i = 0; i < n; i++) {
    h[i] = (float)rand() / RAND_MAX * 100.0f;
  }
}

__global__ void vecAddKernel(float* A, float* B, float* C, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if  (i < n) {
    C[i] = A[i] + B[i];
  }
}

void vecAdd(float* A_h, float* B_h, float* C_h, int n) {
  // Memory Allocation
  int size = n * sizeof(float);
  float* A_d;
  float* B_d;
  float* C_d;

  hipMalloc((void**)&A_d, size);
  hipMalloc((void**)&B_d, size);
  hipMalloc((void**)&C_d, size);

  hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
  hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

  // call kernel code here
  vecAddKernel<<<ceil(n / 32.0), 32>>>(A_d, B_d, C_d, n);
  hipFree(A_d);
  hipFree(B_d);

  hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);
  hipFree(C_d);
}

int main () {
  int n;
  printf("Please enter the size of matrix: ");
  scanf("%d", &n);

  int size = n * sizeof(float);

  float* A_h = (float *)malloc(size);
  float* B_h = (float *)malloc(size);
  float* C_h = (float *)malloc(size);


  genNFP(A_h, n);
  genNFP(B_h, n);

  vecAdd(A_h, B_h, C_h, n);

  free(A_h);
  free(B_h);
  free(C_h);
  return 0;
}